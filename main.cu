#include "hip/hip_runtime.h"
#include <iostream>
#include<random>
#include <algorithm>
#include <thrust/transform.h>
#include <thrust/reduce.h>
#include <thrust/device_vector.h>


using namespace std;


struct Indexes{

    int i;
    int j;
    int size;
};



struct custom_transform
{
    __host__ __device__

    double operator()(const char &a, const char &b)
    { 
        if (a == b)
        {
            return 2;

        }

        return -1;
    }
};




// https://stackoverflow.com/questions/15726641/find-all-possible-substring-in-fastest-way
vector<string> subs_generator(string DNA, int size){
    vector<string> lista_subs;
    
    for(int i = 0; i < size; i++){
        for(int j = i + 1; j < size; j++){

            string x = DNA.substr(i,j);           
            lista_subs.push_back(x);
            
        }
        
        lista_subs.push_back(DNA);
        
    }
    return lista_subs;
}

vector<Indexes> index_generator(string DNA, int size)
{

    vector<Indexes> IndexList;
    Indexes index;

    for (int i = 0; i < size; i++)
    {
        for (int j = i + 1; j < size; j++)
        {
            index.i = i;
            index.j = j;
            index.size = j - i;

            IndexList.push_back(index);
        }
    }


    return IndexList;

}



int main()
{

    int n, m;    
    string SeqA, SeqB;

    cout << "Size Sequence A" << endl;
    cin >> n;
    cout << "Size Sequence B" << endl;
    cin >> m;
    cout << "Sequence A" << endl;
    cin >> SeqA;
    cout << "Sequence B" << endl;
    cin >> SeqB;

    vector<string> all_SeqA;
    vector<string> all_SeqB;

    int max;

    if(n>m){
        max = m;
    }else{
        max = n;
    }

    all_SeqA = subs_generator(SeqA, n);
    all_SeqB = subs_generator(SeqB, m);


    int cand_SeqA, cand_SeqB, id, contador, candidato, Asize, Bsize;
    string finalSeqA, finalSeqB; 
    contador = 0;
    cand_SeqA = 0;
    cand_SeqB = 0;
    int match = 0;




   



    // Removing duplicated SeqA
    std::sort(all_SeqA.begin(), all_SeqA.end());
    all_SeqA.erase(std::unique(all_SeqA.begin(), all_SeqA.end()), all_SeqA.end());

    // Removing duplicated SeqB
    std::sort(all_SeqB.begin(), all_SeqB.end());
    all_SeqB.erase(std::unique(all_SeqB.begin(), all_SeqB.end()), all_SeqB.end());


    vector<Indexes> IndexSeqA = index_generator(SeqA, n);
    vector<Indexes> IndexSeqB = index_generator(SeqB, m);


    vector<char> VSeqA;
    vector<char> VSeqB;

    for(auto&A:SeqA){
    	VSeqA.push_back(A);		
    }
    for(auto&B:SeqB){
    	VSeqB.push_back(B);
    }


    thrust::device_vector<char> SeqA_GPU(VSeqA);
    thrust::device_vector<char> SeqB_GPU(VSeqB);
    thrust::device_vector<int> MatchVec(m);



    for (auto&Seq_A:IndexSeqA)
    {
        for(auto&Seq_B:IndexSeqB)
        {
            if (Seq_A.size == Seq_B.size)
            {
                thrust::transform(SeqA_GPU.begin() + Seq_A.i, SeqA_GPU.begin() + Seq_A.j, SeqB_GPU.begin() + Seq_B.i, MatchVec.begin(), custom_transform());

                int score = thrust::reduce(MatchVec.begin(), MatchVec.end(),0, thrust::plus<int>());
                
                if (score > match)
                {
                    match = score;
                }
            }
        }
    }


    cout << "Resultados Finais:"<< endl;
    cout << "Match Max:"<< endl;
    cout << match << endl;


    cout << ""<< endl;
    cout << "EOF"<< endl;    

    return 0;
}

