#include "hip/hip_runtime.h"
#include <iostream>
#include <algorithm>
#include <thrust/transform.h>
#include <thrust/reduce.h>
#include <thrust/device_vector.h>
#include <vector>
#include <string>




using namespace std;

struct custom_transform
{
    __host__ __device__

    double operator()(const char &a, const char &b)
    { 
        if (a == b)
        {
            return 2;

        }

        return -1;
    }
};




// https://stackoverflow.com/questions/15726641/find-all-possible-substring-in-fastest-way
vector<string> subs_generator(string DNA, int size){
    vector<string> lista_subs;
    
    for(int i = 0; i < size; i++){
        for(int j = i + 1; j < size; j++){

            string x = DNA.substr(i,j);           
            lista_subs.push_back(x);
            
        }
        
        lista_subs.push_back(DNA);
        
    }
    return lista_subs;
}


int main()
{

    int n, m;    
    string SeqA, SeqB;

    cout << "Size Sequence A" << endl;
    cin >> n;
    cout << "Size Sequence B" << endl;
    cin >> m;
    cout << "Sequence A" << endl;
    cin >> SeqA;
    cout << "Sequence B" << endl;
    cin >> SeqB;

    vector<string> all_SeqA;
    vector<string> all_SeqB;

    int max;

    if(n>m){
        max = m;
    }else{
        max = n;
    }

    all_SeqA = subs_generator(SeqA, n);
    all_SeqB = subs_generator(SeqB, m);


    int cand_SeqA, cand_SeqB, id, contador, candidato, Asize, Bsize;
    string finalSeqA, finalSeqB; 
    contador = 0;
    cand_SeqA = 0;
    cand_SeqB = 0;
    int match = 0;
   



    // Removing duplicated SeqA
    std::sort(all_SeqA.begin(), all_SeqA.end());
    all_SeqA.erase(std::unique(all_SeqA.begin(), all_SeqA.end()), all_SeqA.end());

    // Removing duplicated SeqB
    std::sort(all_SeqB.begin(), all_SeqB.end());
    all_SeqB.erase(std::unique(all_SeqB.begin(), all_SeqB.end()), all_SeqB.end());

    thrust::device_vector<char> SeqA_GPU(SeqA);
    thrust::device_vector<char> SeqB_GPU(SeqB);
    thrust::device_vector<int> MatchVec(n);


    for(int i = 0; i < n; i++){
        for(int j = i + 1; j < n; j++){

            Asize = j - i;

            for (int i_B = 0; i_B < m; i_B++) {

                for (int j_B = i_B + 1; j_B < m; j_B++){


                    Bsize = j_B - i_B;

                    if (Asize == Bsize) {

                        thrust::transform(SeqA_GPU.begin() + i, SeqA_GPU.begin() + j, SeqB_GPU.begin() + i_B, MatchVec.begin(), custom_transform());

                        int score = thrust::reduce(MatchVec.begin(), MatchVec.end(),0, thrust::plus<int>());
                        
                        if (score > match) {
                            match = score;
                        }
                    }
                }
            }   
        }
    }
    

    

    cout << "Resultados Finais:"<< endl;
    cout << "Match Max:"<< endl;
    cout << match << endl;


    cout << ""<< endl;
    cout << "EOF"<< endl;    

    return 0;
}
