#include "hip/hip_runtime.h"
#include <iostream>
#include<random>
#include <algorithm>
#include <thrust/transform.h>
#include <thrust/reduce.h>
#include <thrust/device_vector.h>


using namespace std;

struct custom_transform
{
    __host__ __device__

    double operator()(const char& a, const char& b)
    { 
        if (a == b)
        {
            return 2;

        }

        return -1;
    }
};




// https://stackoverflow.com/questions/15726641/find-all-possible-substring-in-fastest-way
vector<string> subs_generator(string DNA, int size){
    vector<string> lista_subs;
    
    for(int i = 0; i < size; i++){
        for(int j = i + 1; j < size; j++){

            string x = DNA.substr(i,j);           
            lista_subs.push_back(x);
            
        }
        
        lista_subs.push_back(DNA);
        
    }
    return lista_subs;
}



int main()
{

    int n, m;    
    string SeqA, SeqB;

    cout << "Size Sequence A" << endl;
    cin >> n;
    cout << "Size Sequence B" << endl;
    cin >> m;
    cout << "Sequence A" << endl;
    cin >> SeqA;
    cout << "Sequence B" << endl;
    cin >> SeqB;

    vector<string> all_SeqA;
    vector<string> all_SeqB;

    int max;

    if(n>m){
        max = m;
    }else{
        max = n;
    }

    all_SeqA = subs_generator(SeqA, n);
    all_SeqB = subs_generator(SeqB, m);


    int cand_SeqA, cand_SeqB, id, contador, candidato, Asize, Bsize;
    string finalSeqA, finalSeqB; 
    contador = 0;
    cand_SeqA = 0;
    cand_SeqB = 0;
    int match = 0;
   



    // Removing duplicated SeqA
    std::sort(all_SeqA.begin(), all_SeqA.end());
    all_SeqA.erase(std::unique(all_SeqA.begin(), all_SeqA.end()), all_SeqA.end());

    // Removing duplicated SeqB
    std::sort(all_SeqB.begin(), all_SeqB.end());
    all_SeqB.erase(std::unique(all_SeqB.begin(), all_SeqB.end()), all_SeqB.end());

    thrust::device_vector<char> SeqA_GPU(n);
    thrust::device_vector<char> SeqB_GPU(m);
    thrust::device_vector<int> MatchVec(m);


for(int r=0; r<n; r++){
	SeqA_GPU[r] = SeqA[r];		
}
	for(int y=0; y<m;y++){
	SeqB_GPU[y] = SeqB[y];
}

    for(int i = 0; i < n; i++){
        for(int j = i + 1; j < n; j++){

            Asize = j - i;

            for (int i_B = 0; i_B < m; i_B++) {

                for (int j_B = i_B + 1; j_B < m; j_B++){


                    Bsize = j_B - i_B;

                    if (Asize == Bsize) {
			cout << SeqA[i];
			cout << SeqB[j];
                        thrust::transform(SeqA_GPU.begin() + i, SeqA_GPU.begin() + j, SeqB_GPU.begin() + i_B, MatchVec.begin(), custom_transform());

                        int score = thrust::reduce(MatchVec.begin(), MatchVec.end(),0, thrust::plus<int>());
                        
                        if (score > match) {
                            match = score;
                        }
                    }
                }
            }   
        }
    }
    

    

    cout << "Resultados Finais:"<< endl;
    cout << "Match Max:"<< endl;
    cout << match << endl;


    cout << ""<< endl;
    cout << "EOF"<< endl;    

    return 0;
}

